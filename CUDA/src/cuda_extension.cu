#include "cuda_extension.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

namespace cuda_extension {

hipblasHandle_t handle;

inline int initDevice(IN_Layer inLayer, std::vector<NN_Layer> hiddenLayers, NN_Layer outLayer) {

	hipblasStatus_t status;
	hipError_t err;
	const char *str = "";

	// Get CUDA device
	int dev = findCudaDevice(0, &str);
	if (dev == -1) {
		return EXIT_FAILURE;
	}

	// Create CUBLAS Handle
	printf("INITIALIZING CUBLASS....");

	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "CUBLASS FAILED TO CREATE HANDLE!");
		return EXIT_FAILURE;
	}

	printf("CUBLASS INITIALIZED!");


	// Allocate Memory on device for layers
	printf("ALLOCATING DEVICE MEMORY...");

	err = hipMalloc(reinterpret_cast<void**>(&(inLayer.inputs)), inLayer.Nodes * sizeof(float));

	if( err != hipSuccess) {
		fprintf(stderr, "DEVICE MEMORY ALLOCATION ERROR: UNABLE TO ALLOCATE INPUT LAYER");
		return EXIT_FAILURE;
	}

	err = hipMalloc(reinterpret_cast<void**>(&(outLayer.output)), outLayer.Nodes * sizeof(float));

	if( err != hipSuccess) {
		fprintf(stderr, "DEVICE MEMORY ALLOCATION ERROR: UNABLE TO ALLOCATE OUTPUT LAYER");
		return EXIT_FAILURE;
	}

	err = hipMalloc(reinterpret_cast<void**>(&(outLayer.bias)), outLayer.Nodes * sizeof(float));

	if( err != hipSuccess) {
		fprintf(stderr, "DEVICE MEMORY ALLOCATION ERROR: UNABLE TO ALLOCATE OUTPUT LAYER");
		return EXIT_FAILURE;
	}

	err = hipMalloc(reinterpret_cast<void**>(&(outLayer.error)), outLayer.Nodes * sizeof(float));

	if( err != hipSuccess) {
		fprintf(stderr, "DEVICE MEMORY ALLOCATION ERROR: UNABLE TO ALLOCATE OUTPUT LAYER");
		return EXIT_FAILURE;
	}

	err = hipMalloc(reinterpret_cast<void**>(&(outLayer.weights)), *outLayer.wRows * outLayer.wCols * sizeof(float));

	if( err != hipSuccess) {
		fprintf(stderr, "DEVICE MEMORY ALLOCATION ERROR: UNABLE TO ALLOCATE OUTPUT LAYER");
		return EXIT_FAILURE;
	}


	for (auto i : hiddenLayers) {
		err = hipMalloc(reinterpret_cast<void**>(&(i.output)), i.Nodes * sizeof(float));

		if( err != hipSuccess) {
			fprintf(stderr, "DEVICE MEMORY ALLOCATION ERROR: UNABLE TO ALLOCATE HIDDEN LAYER %d", i);
			return EXIT_FAILURE;
		}

		err = hipMalloc(reinterpret_cast<void**>(&(i.bias)), i.Nodes * sizeof(float));

		if( err != hipSuccess) {
			fprintf(stderr, "DEVICE MEMORY ALLOCATION ERROR: UNABLE TO ALLOCATE OUTPUT LAYER");
			return EXIT_FAILURE;
		}

		err = hipMalloc(reinterpret_cast<void**>(&(i.error)), i.Nodes * sizeof(float));

		if( err != hipSuccess) {
			fprintf(stderr, "DEVICE MEMORY ALLOCATION ERROR: UNABLE TO ALLOCATE OUTPUT LAYER");
			return EXIT_FAILURE;
		}

		err = hipMalloc(reinterpret_cast<void**>(&(i.weights)), *i.wRows * i.wCols * sizeof(float));

		if( err != hipSuccess) {
			fprintf(stderr, "DEVICE MEMORY ALLOCATION ERROR: UNABLE TO ALLOCATE OUTPUT LAYER");
			return EXIT_FAILURE;
		}
	}

	printf("MEMORY ALLOCATED!");

	return 0;



}




}
