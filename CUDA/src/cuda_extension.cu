#include "hip/hip_runtime.h"
#include "cuda_extension.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define  CUDA_CALL(x) do { if((x) !=  hipSuccess) { \
	printf ("Error  at %s:%d\n",__FILE__ ,__LINE__); \
	return  EXIT_FAILURE ;}}  while (0)

namespace cuda_extension {

hipblasHandle_t handle;

inline int initDevice(IN_Layer inLayer, std::vector<NN_Layer> hiddenLayers, NN_Layer outLayer) {

	hipblasStatus_t status;
	hipError_t err;
	const char *str = "";

	hiprandState *devStates;

	// Get CUDA device
	int dev = findCudaDevice(0, &str);
	if (dev == -1) {
		return EXIT_FAILURE;
	}

	// Create CUBLAS Handle
	printf("INITIALIZING CUBLASS...");

	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "CUBLASS FAILED TO CREATE HANDLE!");
		return EXIT_FAILURE;
	}

	printf("CUBLASS INITIALIZED!");


	// Allocate Memory on device for layers
	printf("ALLOCATING DEVICE MEMORY...");

	// Input Layer
	CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&(inLayer.inputs)), inLayer.Nodes * sizeof(float)));

	// Output Layer
	CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&(outLayer.output)), outLayer.Nodes * sizeof(float)));

	CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&(outLayer.bias)), outLayer.Nodes * sizeof(float)));

	CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&(outLayer.error)), outLayer.Nodes * sizeof(float)));

	CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&(outLayer.weights)), *outLayer.wRows * outLayer.wCols * sizeof(float)));

	// Hidden Layers
	for (auto i : hiddenLayers) {
		CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&(i.output)), i.Nodes * sizeof(float)));

		CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&(i.bias)), i.Nodes * sizeof(float)));

		CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&(i.error)), i.Nodes * sizeof(float)));

		CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&(i.weights)), *i.wRows * i.wCols * sizeof(float)));

	}

	printf("MEMORY ALLOCATED!");

//	// Fill device weight and bias matrices with random data with CURAND
//	printf("FILLING WEIGHTS AND BIAS WITH RANDOM VALUES...");
//
//	printf("INITIALIZING CURAND...");
//
//	CUDA_CALL(hipMalloc(reinterpret_cast<void**>(&devStates), sizeof(hiprandState)));
//
//
	return 0;


}

//__global__
//void deviceRandomFill (float *matrix, unsigned size, hiprandState *globalState) {
//	hiprandState localState;
//	localState = global_State[]
//}



}
