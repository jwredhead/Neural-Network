#include "hip/hip_runtime.h"
// CUSTOM INCLUDES
#include "cuda_extension.h"


// CPP INCLUDES
#include <stdio.h>
#include <stdlib.h>
#include <map>
#include <string>

// CUDA INCLUDES
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"

// Macro for output of cuda errors
#define  CUDA_CALL(x) do { if((x) !=  hipSuccess) { \
	printf ("Error  at %s:%d\nError Value: %s\n",__FILE__ ,__LINE__, hipGetErrorString(x)); \
	exit(EXIT_FAILURE);}}  while (0)

namespace cuda_extension {

// Namespace-wide variables
hipblasHandle_t handle; // Handle for Cublas
unsigned maxNodes; // Max number of nodes


// Initialization kernel for GPU Random Number Generator
__global__ void initRand (hiprandState_t *state) {
	int id = threadIdx.x + blockIdx.x * 256;

	hiprand_init(1234, id, 0, &state[id]);
}

// Kernel to fill a given matrix with random numbers
__global__ void deviceRandomFill (float *matrix, unsigned size, hiprandState_t *globalState) {

	int id = threadIdx.x + blockIdx.x * 256;

	hiprandState_t localState;
	localState = globalState[id];

	float s_value = hiprand_uniform(&localState);

	__syncthreads();

	if(id < size) {
		matrix[id] = s_value;
	}
	globalState[id] = localState;
}

// Kernel to calculate the error between a target matrix and an error matrix
__global__ void calcError (float *targets, float *output, float *error, unsigned size) {
	int id = threadIdx.x + blockIdx.x * 256;

	float t = 0, e = 0, o = 0;

	if (id < size) {
		t = targets[id];
		e = error[id];
		o = output[id];
	}
	__syncthreads();

	e = t - o;

	__syncthreads();

	if (id < size) {
		error[id] = e;
	}
}

// Kernel to add 2 matrices
__global__ void matrixAdd (float *x, float *y, unsigned size) {
	int id = threadIdx.x + blockIdx.x * 256;

	float  s_x = 0, s_y = 0;

	if (id < size) {
		s_x = x[id];
		s_y = y[id];
	}

	__syncthreads();

	s_x += s_y;

	__syncthreads();

	if(id < size) {
		x[id] = s_x;
	}
}

// Kernel to run sigmoid function over every element of a matrix
__global__ void runSigmoidFunction(float *x, unsigned size) {
	int id = threadIdx.x + blockIdx.x * 256;

	float t = 0;

	if (id < size) {
		t = x[id];
	}

	__syncthreads();

	t = (1 - expf(-t) / (1 + expf(-t)));

	__syncthreads();

	if (id < size) {
		x[id] = t;
	}
}

// Kernel to run bisigmoid function over every element of a matrix
__global__ void runBisigmoidFunction(float *x, unsigned size) {
	int id = threadIdx.x + blockIdx.x * 256;

	float t = 0;

	if (id < size) {
		t = x[id];
	}

	__syncthreads();

	t = 1 - expf(-t) / (1 + expf(-t));

	__syncthreads();

	if (id < size) {
		x[id] = t;
	}
}

// Kernel to run tanh function over every element of a matrix
__global__ void runTanhFunction(float *x, unsigned size) {
	int id = threadIdx.x + blockIdx.x * 256;

	float t = 0;

	if (id < size) {
		t = x[id];
	}

	__syncthreads();

	t = tanh(t);

	__syncthreads();

	if (id < size) {
		x[id] = t;
	}
}

// Kernel to calculate the gradient between outputs, x, and errors, y,
__global__ void calcGradientSigmoid( float *gradient, float *x, float *y, unsigned size, float lr) {
	int id = threadIdx.x + blockIdx.x * 256;

	float t = 0, e = 0, g = 0;
	if (id < size) {
		t = x[id];
		e = y[id];
	}

	__syncthreads();

	g = t * ( 1 - t);
	g = g * e;
	g = g * lr;

	__syncthreads();

	if (id < size) {
		gradient[id] = g;
	}
}

// Kernel to calculate the gradient between outputs, x, and errors, y,
__global__ void calcGradientBisigmoid( float *gradient, float *x, float *y, unsigned size, float lr) {
	int id = threadIdx.x + blockIdx.x * 256;

	float t = 0, e = 0, g = 0;
	if (id < size) {
		t = x[id];
		e = y[id];
	}

	__syncthreads();

	g = 2 * t * (1 - t);
	g = g * e;
	g = g * lr;

	__syncthreads();

	if (id < size) {
		gradient[id] = g;
	}
}

// Kernel to calculate the gradient between outputs, x, and errors, y,
__global__ void calcGradientTanh( float *gradient, float *x, float *y, unsigned size, float lr) {
	int id = threadIdx.x + blockIdx.x * 256;

	float t = 0, e = 0, g = 0;
	if (id < size) {
		t = x[id];
		e = y[id];
	}

	__syncthreads();

	g = 1 - t * t;
	g = g * e;
	g = g * lr;

	__syncthreads();

	if (id < size) {
		gradient[id] = g;
	}
}
// Convert MATRIX_OP to hipblasOperation_t
inline hipblasOperation_t convertToCublasOp (MATRIX_OP t) {

	hipblasOperation_t s;

	std::map<MATRIX_OP, hipblasOperation_t> map = {
			{MATRIX_OP::NORMAL, hipblasOperation_t::HIPBLAS_OP_N},
			{MATRIX_OP::TRANSPOSE, hipblasOperation_t::HIPBLAS_OP_N}
	};

	s = map.at(t);

	return s;
}

// If device ptr isn't null, free it
inline int cudaDelete (float *a) {
	if (a) {
		CUDA_CALL(hipFree(a));
	}
	return 0;
}

// Initialize Neural Network Layers on the device
void initLayers(IN_Layer *inLayer, std::vector<NN_Layer> *hiddenLayers, NN_Layer *outLayer) {

	hipblasStatus_t status;
	const char *str = "";

	hiprandState_t *devStates;

	// Get CUDA device
	int dev = findCudaDevice(0, &str);
	if (dev == -1) {
		fprintf(stderr, "FINDCUDADEVICE ERROR: -1\n");
		exit(EXIT_FAILURE);
	}

	// Create CUBLAS Handle
	printf("INITIALIZING CUBLASS...\n");

	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "CUBLASS FAILED TO CREATE HANDLE!\nERROR: %s\n", _cudaGetErrorEnum(status));
		exit(EXIT_FAILURE);
	}

	printf("CUBLASS INITIALIZED!\n");


	// Allocate Memory on device for layers
	printf("ALLOCATING DEVICE MEMORY...\n");

	// Input Layer
	CUDA_CALL(hipMalloc(&(inLayer->inputs), inLayer->Nodes * sizeof(float)));

	maxNodes = inLayer->Nodes;

	// Output Layer
	CUDA_CALL(hipMalloc(&(outLayer->output), outLayer->Nodes * sizeof(float)));

	CUDA_CALL(hipMalloc(&(outLayer->bias), outLayer->Nodes * sizeof(float)));

	CUDA_CALL(hipMalloc(&(outLayer->error), outLayer->Nodes * sizeof(float)));

	CUDA_CALL(hipMalloc(&(outLayer->weights), outLayer->wRows * outLayer->wCols * sizeof(float)));

	if (outLayer->Nodes > maxNodes) {
		maxNodes = outLayer->Nodes;
	}

	// Hidden Layers
	for (auto&& i : *hiddenLayers) {
		CUDA_CALL(hipMalloc(&(i.output), i.Nodes * sizeof(float)));

		CUDA_CALL(hipMalloc(&(i.bias), i.Nodes * sizeof(float)));

		CUDA_CALL(hipMalloc(&(i.error), i.Nodes * sizeof(float)));

		CUDA_CALL(hipMalloc(&(i.weights), i.wRows * i.wCols * sizeof(float)));

		if (i.Nodes > maxNodes) {
			maxNodes = i.Nodes;
		}

	}

	printf("MEMORY ALLOCATED!\n");

	// Fill device weight and bias matrices with random data with CURAND
	printf("FILLING WEIGHTS AND BIAS WITH RANDOM VALUES...\n");

	printf("INITIALIZING CURAND...\n");

	unsigned numBlocks = ceil(maxNodes * maxNodes / MAX_THREADS_PER_BLOCK);
	unsigned numThreads = (maxNodes > MAX_THREADS_PER_BLOCK) ? MAX_THREADS_PER_BLOCK : maxNodes;
	dim3 dimGrid(numBlocks, 1, 1);
	dim3 dimBlock(numThreads, 1, 1);

	CUDA_CALL(hipMalloc(&devStates, numBlocks * MAX_THREADS_PER_BLOCK * sizeof(hiprandState_t)));

	initRand<<<dimGrid,dimBlock>>>(devStates);

	printf("CURAND INITIALIZED!\n");

	deviceRandomFill<<<dimGrid, dimBlock>>>(outLayer->weights, outLayer->wRows * outLayer->wCols, devStates);
	deviceRandomFill<<<dimGrid, dimBlock>>>(outLayer->bias, outLayer->Nodes, devStates);

	for (auto i : *hiddenLayers) {
		deviceRandomFill<<<dimGrid, dimBlock>>>(i.weights, i.wRows * i.wCols, devStates);
		deviceRandomFill<<<dimGrid, dimBlock>>>(i.bias, i.Nodes, devStates);
	}

	CUDA_CALL(hipDeviceSynchronize());
	printf("WEIGHT AND BIAS MATRICES FILLED!\n");

}

// Delete Neural Network from device
void deleteLayers(IN_Layer *inLayer, std::vector<NN_Layer> *hiddenLayers, NN_Layer *outLayer) {
	printf("DELETING LAYERS FROM DEVICE...\n");
	int error;

	error = cudaDelete(inLayer->inputs);

	error += cudaDelete(outLayer->weights);
	error +=cudaDelete(outLayer->bias);
	error +=cudaDelete(outLayer->error);
	error +=cudaDelete(outLayer->output);

	for ( auto&& i : *hiddenLayers) {
		error +=cudaDelete(i.weights);
		error +=cudaDelete(i.bias);
		error +=cudaDelete(i.error);
		error +=cudaDelete(i.output);
	}
	if (error ==0) {
		printf("LAYERS DELETED FROM DEVICE!\n");
	} else {
		printf("UNABLE TO DELETE LAYERS FROM DEVICE!/nNUMBER OF ERRORS: %d\n", error);
	}

	printf("CLOSING CUBLAS...\n");
	hipblasStatus_t status = hipblasDestroy(handle);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "CUBLASS FAILED TO DESTROY HANDLE!\nERROR: %s\n", _cudaGetErrorEnum(status));
	} else
	{
		printf("CUBLASS CLOSED!\n");
	}

}

// Wrapper for CUBLAS SGEMM matrix multiplication, calculates C = A * B + C
void multiplyAccumulate(MATRIX_OP transA, float *A, MATRIX_OP transB, float *B, float *C, int A_rows, int A_cols, int B_cols) {
	float alpha = 1.0;
	float beta = 1.0;


	hipblasStatus_t status;
	status = hipblasSgemm(handle,convertToCublasOp(transB), convertToCublasOp(transA), B_cols, A_rows, A_cols, &alpha, B, B_cols, A, A_cols, &beta, C, B_cols);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "CUBLASS MATRIX MULTIPLY ERROR: %s\n", _cudaGetErrorEnum(status));
	}
}

// Wrapper for CUBLAS SGEMM matrix multiplication, calculates C = A * B
void multiply(MATRIX_OP transA, float *A, MATRIX_OP transB, float *B, float *C, int A_rows, int A_cols, int B_cols) {
	float alpha= 1.0;
	float beta = 0.0;


	hipblasStatus_t status;
	status = hipblasSgemm(handle,convertToCublasOp(transB), convertToCublasOp(transA), B_cols, A_rows, A_cols, &alpha, B, B_cols, A, A_cols, &beta, C, B_cols);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "CUBLASS MATRIX MULTIPLY ERROR: %s\n", _cudaGetErrorEnum(status));
	}
}

// Copies data from src to dst, where both pointers are device ptrs
void copyVector(NN_Layer *layer) {
	CUDA_CALL(hipMemcpy(layer->output, layer->bias,  layer->Nodes * sizeof(float), hipMemcpyDeviceToDevice));
}

// Copies inputs over to device
void copyInputs(float *src, IN_Layer *inLayer) {
	CUDA_CALL(hipMemcpy(inLayer->inputs, src, inLayer->Nodes * sizeof(float), hipMemcpyHostToDevice));
}

// Runs activation function over matrix
void activationFunction(float *x, unsigned size, Activation_Function f) {

	unsigned numBlocks = ceil(maxNodes * maxNodes / MAX_THREADS_PER_BLOCK);
	unsigned numThreads = (maxNodes > MAX_THREADS_PER_BLOCK) ? MAX_THREADS_PER_BLOCK : maxNodes;
	dim3 dimGrid(numBlocks, 1, 1);
	dim3 dimBlock(numThreads, 1, 1);
	switch (f) {
		case Activation_Function::SIGMOID: runSigmoidFunction<<<dimGrid, dimBlock>>>(x, size); break;
		case Activation_Function::BI_SIGMOID: runBisigmoidFunction<<<dimGrid, dimBlock>>>(x, size); break;
		case Activation_Function::TANH: runTanhFunction<<<dimGrid, dimBlock>>>(x, size); break;
	}

	CUDA_CALL(hipDeviceSynchronize());
}

// Calculates the error between given targets and outputs
void calculateError(float *targets, float *outputs, float *error, unsigned size) {
	float * d_targets;
	CUDA_CALL(hipMalloc(&d_targets, size * sizeof(float)));
	CUDA_CALL(hipMemcpy(d_targets, targets, size * sizeof(float), hipMemcpyHostToDevice));


	unsigned numBlocks = ceil(maxNodes * maxNodes / MAX_THREADS_PER_BLOCK);
	unsigned numThreads = (maxNodes > MAX_THREADS_PER_BLOCK) ? MAX_THREADS_PER_BLOCK : maxNodes;
	dim3 dimGrid(numBlocks, 1, 1);
	dim3 dimBlock(numThreads, 1, 1);

	calcError<<<dimGrid, dimBlock>>>(d_targets, outputs, error, size);

	CUDA_CALL(hipDeviceSynchronize());
}

// Adjusts a given layer's weights and bias back propogating gradient descent
void adjustWeightsBias(NN_Layer *layer, float *inputs, unsigned inputSize, Activation_Function f, float learningRate) {

	float *gradient, *deltaWeight;

	CUDA_CALL(hipMalloc(&gradient, layer->Nodes * sizeof(float)));
	CUDA_CALL(hipMalloc(&deltaWeight, layer->Nodes * inputSize *sizeof(float)));

	unsigned numBlocks = ceil(maxNodes * maxNodes / MAX_THREADS_PER_BLOCK);
	unsigned numThreads = (maxNodes > MAX_THREADS_PER_BLOCK) ? MAX_THREADS_PER_BLOCK : maxNodes;
	dim3 dimGrid(numBlocks, 1, 1);
	dim3 dimBlock(numThreads, 1, 1);

	switch (f) {
		case Activation_Function::SIGMOID: calcGradientSigmoid<<<dimGrid, dimBlock>>>(gradient, layer->output, layer->error, layer->Nodes, learningRate); break;
		case Activation_Function::BI_SIGMOID: calcGradientBisigmoid<<<dimGrid, dimBlock>>>(gradient, layer->output, layer->error, layer->Nodes, learningRate); break;
		case Activation_Function::TANH: calcGradientTanh<<<dimGrid, dimBlock>>>(gradient, layer->output, layer->error, layer->Nodes, learningRate); break;
	}

	CUDA_CALL(hipDeviceSynchronize());

	multiply(MATRIX_OP::NORMAL, gradient, MATRIX_OP::TRANSPOSE, inputs, deltaWeight, layer->Nodes, 1, inputSize);

	matrixAdd<<<dimGrid, dimBlock>>>(layer->weights, deltaWeight, layer->Nodes);
	matrixAdd<<<dimGrid, dimBlock>>>(layer->bias, gradient, layer->Nodes);

	CUDA_CALL(hipDeviceSynchronize());

	cudaDelete(gradient);
	cudaDelete(deltaWeight);
}

// Retrieves Neural Network outputs from device to host
void getOutputs(float *d_outputs, float *h_outputs, unsigned size) {

	CUDA_CALL(hipMemcpy(reinterpret_cast<void*>(h_outputs), reinterpret_cast<void*>(d_outputs), size * sizeof(float), hipMemcpyDeviceToHost));
}

} /* namespace cuda_extension */

#undef CUDA_CALL
